

#include <hip/hip_runtime.h>
#include <iostream>
#include <assert.h>

#define	H 3
#define	W 5

inline hipError_t checkCuda(hipError_t result)
{
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
  }
  return result;
}

__global__	void	fire(float *input, float *weight, size_t pitch, float *res) {
	int	i = threadIdx.x;

	if (i < H) {
		for (int idx = 0; idx < W; idx++) {
			float	*row = (float*)((char*)weight + (i * pitch));
			res[i] += input[idx] * row[idx];
		}
	}
}

int main(void) {

	float	*h_test = NULL;
	float	*h_alpha = NULL;
	float	*h_result = NULL;

	checkCuda( hipHostMalloc((void**)&h_test,W*H*sizeof(float), hipHostMallocDefault) );
	float	num = 0;
	for (int i = 0; i < W * H; i++, num++) h_test[i] = num;

	checkCuda( hipHostMalloc((void**)&h_alpha, W*sizeof(float), hipHostMallocDefault) );
	num = 0;
	for (int i = 0; i < H; i++, num++) h_alpha[i] = num;

	checkCuda( hipHostMalloc((void**)&h_result, H*sizeof(float), hipHostMallocDefault) );

	float	*d_test = NULL;
	float	*d_alpha = NULL;
	float	*d_result = NULL;

	size_t	pitch_test = 0;
	checkCuda( hipMallocPitch((void**)&d_test, &pitch_test, W*sizeof(float), H) );
	checkCuda( hipMemcpy2D(d_test, pitch_test, h_test, W*sizeof(float), W*sizeof(float), H, hipMemcpyHostToDevice) );

	checkCuda( hipMalloc((void**)&d_alpha, W*sizeof(float)) );
	checkCuda( hipMemcpy(d_alpha, h_alpha, W*sizeof(float), hipMemcpyHostToDevice) );

	checkCuda( hipMalloc((void**)&d_result, H*sizeof(float)) );
	checkCuda( hipMemset(d_result, 0, H*sizeof(float)) );

	fire<<<1, H>>>(d_alpha, d_test, pitch_test, d_result);

	checkCuda( hipMemcpy(h_result, d_result, H*sizeof(float), hipMemcpyDeviceToHost) );

	printf("result :");
	for (int i = 0; i < H; i++) {
		printf(" %f ", h_result[i]);
	}
	printf("\n");

	hipFree(d_alpha);
	hipFree(d_test);
	hipFree(d_result);
	hipHostFree(h_alpha);
	hipHostFree(h_test);
	hipHostFree(h_result);
	return 0;
}