

#include <hip/hip_runtime.h>
#include <stdio.h>
#define	SIZE 100


__global__ void	fire(float *test) {
	int const	i = threadIdx.x;

	if (i < SIZE) test[i] += 1.; 
}

int	main( void ) {
	float	*h_array;
	float	*d_array;

	unsigned int const	bytes = SIZE * sizeof(float);

	hipHostMalloc((void**)&h_array, bytes, hipHostMallocDefault);
	hipMalloc((void**)&d_array, bytes);

	for (int idx = 0; idx < SIZE; idx++) h_array[idx] = (float)idx;

	hipMemcpy(d_array, h_array, bytes, hipMemcpyHostToDevice);

	hipEvent_t	startEvent, stopEvent;

	hipEventCreate(&startEvent);
	hipEventCreate(&stopEvent);

	hipEventRecord(startEvent, 0);
	fire<<<1, SIZE>>>(d_array);
	hipEventRecord(stopEvent, 0);
	hipEventSynchronize(stopEvent);

	float time;
	hipEventElapsedTime(&time, startEvent, stopEvent);
	printf("  KernelCall (ms): %f\n", time);

	hipFree(d_array);
	hipHostFree(h_array);

	return 0;
}