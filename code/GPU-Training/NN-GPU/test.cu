

#include <hip/hip_runtime.h>
#include <iostream>

int main(void) {

	int value;
	int device;
	
	hipGetDevice(&device);
	hipDeviceGetAttribute(&value, hipDeviceAttributeL2CacheSize, device);

	std::cout << "number of SMs: " << value << " on device: " << device << std::endl;
	return 0;
}