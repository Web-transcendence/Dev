
#include <iostream>

#define	H 3
#define	W 5

int main(void) {

	float	*h_test;

	hipHostMalloc((void**)h_test,W * H * sizeof(float));
	
	float	num = 0;
	for (int i = 0; i < W * H; i++) h_test[i] = num;

	float	**d_test;
	size_t	pitch;

	hipMallocPitch((void**)d_test, &pitch, W * sizeof(float), H);
	
	hipMemcpy2D();
	return 0;
}