#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   Layer.cpp                                          :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: thibaud <thibaud@student.42.fr>            +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2025/03/16 14:04:30 by thibaud           #+#    #+#             */
/*   Updated: 2025/04/01 10:56:01 by thibaud          ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "Layer.class.hpp"
#include "Neuron.class.hpp"
#include "Math.namespace.hpp"

Layer::Layer(int const n_neurons, int const n_weights, t_actFunc actFunc) :\
	sizeNeurons(n_neurons), sizeWeight(n_weights) {
	hipError_t	errN[3];
	errN[0] = hipMalloc(&this->weight, n_neurons * sizeof(double*));
	errN[1] = hipMalloc(&this->nablaW, n_neurons * sizeof(double*));
	errN[2] = hipMalloc(&this->deltaNablaW, n_neurons * sizeof(double*));
	if (errN[0] != hipSuccess || errN[1] != hipSuccess || errN[2] != hipSuccess)
		throw cudaMallocException();
	for (int i = 0; i < n_neurons; i++) {
		errN[0] = hipMalloc(&this->weight[i], n_weights * sizeof(double));
		errN[1] = hipMalloc(&this->nablaW , n_weights * sizeof(double));
		errN[2] = hipMalloc(&this->deltaNablaW , n_weights * sizeof(double));
		if (errN[0] != hipSuccess || errN[1] != hipSuccess || errN[2] != hipSuccess)
			throw cudaMallocException();
	}
	// il faut instancier les weights et biaises
	errN[0] = hipMalloc(&this->biais, n_weights * sizeof(double));
	errN[1] = hipMalloc(&this->nablaB, n_weights * sizeof(double));
	errN[2] = hipMalloc(&this->deltaNablaB, n_weights * sizeof(double));
	if (errN[0] != hipSuccess || errN[1] != hipSuccess || errN[2] != hipSuccess)
			throw cudaMallocException();
	this->_actFuncSingle = Math::actFuncS[actFunc];
	this->_actFuncVector = Math::actFuncV[actFunc];
	this->_primeActFuncSingle = Math::primeActFuncS[actFunc];
	this->_primeActFuncVector = Math::primeActFuncV[actFunc];
	return ;
}

Layer::~Layer( void ) {
	if (this->weight && this->nablaW && this->deltaNablaW) {
		for (int i = 0; i < this->sizeNeurons; i++) {
			if (this->weight[i]) {hipFree(this->weight[i]);}
			if (this->nablaW[i]) {hipFree(this->nablaW[i]);}
			if (this->deltaNablaW[i]) {hipFree(this->deltaNablaW[i]);}
		}	
	}
	if (weight) {hipFree(this->weight);}
	if (nablaW) {hipFree(this->nablaW);}
	if (deltaNablaW) {hipFree(this->deltaNablaW);}
	if (biais) {hipFree(this->biais);}
	if (nablaB) {hipFree(this->nablaB);}
	if (deltaNablaB) {hipFree(this->deltaNablaB);}
	return ;
}

__global__ void	fireFeedForward(double const *input, double const **weight, double const *bias, double *res, ptrFuncS funcPtr, int const size) {
	int const	i = threadIdx.x;

	Math::dotProduct(input, weight[i], &res[i], size);
	res[i] += bias[i];
	(funcPtr)(&res[i]);
	return ;
}

double	*Layer::feedForward(double const *input) {
	double		*res;
	hipError_t	err;

	err = hipMalloc(&res, this->sizeNeurons * sizeof(double));
	if (err != hipSuccess)
		throw	cudaMallocException();
	fireFeedForward<<<1, this->sizeNeurons>>>(input, this->weight, this->biais, res, this->_actFuncSingle, this->sizeNeurons);
	return res;
}

__global__ void	fireAffineTransformation(double const *input, double const **weight, double const *bias, double *res, ptrFuncS funcPtr, int const size) {
	int const	i = threadIdx.x;

	Math::dotProduct(input, weight[i], &res[i], size);
	res[i] += bias[i];
	return ;
}

double*	Layer::affineTransformation(double const *input) {
	double		*res;
	hipError_t	err;

	err = hipMalloc(&res, this->sizeNeurons * sizeof(double));
	if (err != hipSuccess)
		throw	cudaMallocException();
	fireAffineTransformation<<<1, this->sizeNeurons>>>(input, this->weight, this->biais, res, this->_actFuncSingle, this->sizeNeurons);;
	return res;
}

double	Layer::callActFunc(double const input) {
	return this->_actFuncSingle(input);
}

std::vector<double>*	Layer::callActFunc(std::vector<double> const & input) {
	return this->_actFuncVector(input);
}

double	Layer::callPrimeActFunc(double const input) {
	return this->_primeActFuncSingle(input);
}

std::vector<double>*	Layer::callPrimeActFunc(std::vector<double> const & input) {
	return this->_primeActFuncVector(input);
}

void	Layer::updateWeight(double const eta, double const miniBatchSize) {
	for (auto it_n = this->_neurons.begin(); it_n != this->_neurons.end(); it_n++)
		(*it_n)->updateWeight(eta, miniBatchSize);
	return ;
}

void	Layer::updateNabla_w( void ) {
	for (auto it_n = this->_neurons.begin(); it_n != this->_neurons.end(); it_n++)
		(*it_n)->updateNabla_w();
	return ;
}

void	Layer::setDeltaNabla_w(std::vector<double> const & delta, std::vector<double> const & activation) {
	auto	product = Math::outerProduct(delta, activation);
	auto	it_p = product->begin();
	
	for (auto it_n = this->_neurons.begin(); it_n != this->_neurons.end(); it_n++, it_p++)
		(*it_n)->setDeltaNabla_w(*it_p);
	delete product;
	return ;
}

void	Layer::updateBias(double const eta, double const miniBatchSize) {
	for (auto it_n = this->_neurons.begin(); it_n != this->_neurons.end(); it_n++)
		(*it_n)->updateBias(eta, miniBatchSize);
	return ;
}

void	Layer::updateNabla_b( void ) {
	for (auto it_n = this->_neurons.begin(); it_n != this->_neurons.end(); it_n++)
		(*it_n)->updateNabla_b();
	return ;
}

void	Layer::setDeltaNabla_b(std::vector<double> const & delta) {
	auto	it_n = this->_neurons.begin();
	auto	it_d = delta.begin();
	for (;it_d != delta.end() && it_n != this->_neurons.end(); it_d++, it_n++) {
		(*it_n)->setDeltaNabla_b(*it_d);
	}
	return ;
}

std::vector<double>*	Layer::calcDelta(std::vector<double> const & delta, std::vector<double> const & sp) {
	auto	merged = std::vector<std::vector<double>>(this->_neurons.size(), std::vector<double>(this->_neurons.at(0)->_weight.size()));
	auto	it = merged.begin();

	for (auto it_n = this->_neurons.begin(); it_n != this->_neurons.end(); it_n++) {
		auto	it_w = (*it).begin();
		for (auto it_we = (*it_n)->_weight.begin(); it_we != (*it_n)->_weight.end(); it_we++) {
			*it_w = *it_we;
			++it_w;
		}
		++it;
	}
	auto	transposed = Math::transpose2D(merged);
	auto	temp = std::vector<double>(this->_neurons.at(0)->_weight.size());
	auto	it_t = temp.begin();
	for (auto it_tr = transposed->begin(); it_tr != transposed->end(); it_tr++, it_t++) {
		(*it_t) = Math::dotProduct(*it_tr, delta);
	}
	delete transposed;
	return Math::hadamardProduct(temp, sp);	
}
